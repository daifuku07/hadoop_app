#include "hip/hip_runtime.h"
/********************
* @Author: Peilong Li
********************/
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
// includes, project
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include "Main.h"

// includes, kernels
//#include <matrixAdd_kernel.cu>
#ifndef _MATRIXADD_KERNEL_H_
#define _MATRIXADD_KERNEL_H_

#include <stdio.h>

#define SDATA( index)      cutilBankChecker(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Kernel that executes on the CUDA device
#ifdef __cplusplus
extern "C"
{
#endif
    
__global__ void add_matrix(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];

}


__global__ void mmul_matrix(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] * b[idx];
}
#ifdef __cplusplus
}
#endif 
#endif // #ifndef _MATRIXADD_KERNEL_H_
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
/*
 * void runTest( int argc, char** argv);
 *
 * extern "C"
 * void computeGold( float* reference, float* idata, const unsigned int len);
 */
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

#ifdef __cplusplus
extern "C"
{
#endif
	// CUDA code here
	JNIEXPORT jint JNICALL Java_Main_CUDAProxy_1matrixMmul(JNIEnv *env, jobject obj, jfloatArray aArray, jfloatArray bArray, jfloatArray cArray){
		printf("C: fetching arrays from Java\n");
		jfloat *a_h = env->GetFloatArrayElements(aArray, 0);
		jfloat *b_h = env->GetFloatArrayElements( bArray, 0);
		jfloat *c_h = env->GetFloatArrayElements( cArray, 0);
		printf("C: Got reference to all a, b, and c\n");
		jsize N = 3;
		printf("C: calling CUDA kernel\n");
		
		float *a_d, *b_d, *c_d;
		//const int N = 10;

		size_t size = N * sizeof (float);
		
		// allocate memory in the GPU device for a, b and c
		hipMalloc((void **) & a_d, size);
		hipMalloc((void **) & b_d, size);
		hipMalloc((void **) & c_d, size);
		// copy from host to GPU device
		hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
		hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
		// do calculations on device
		int block_size = 4;
		int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
		add_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
		// Retrieve results from the device
		hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
		// print out the results
		printf("CU: c[]:");
		for (int i = 0; i < N; i++) printf("%5.2f|", c_h[i]);
		printf("\n");
		
		hipFree(a_d);
		hipFree(b_d);
		hipFree(c_d);
		
		printf("C: back from CUDA kernel, coping data to Java\n");
		env->ReleaseFloatArrayElements(aArray, a_h, 0);
		env->ReleaseFloatArrayElements(bArray, b_h, 0);
		env->ReleaseFloatArrayElements(cArray, c_h, 0);
		printf("C: Going back to Java\n");
		return (jint) N; // this might not be the right way to return values to Java
	}


#ifdef __cplusplus
}
#endif 
